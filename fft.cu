#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

#define	R2	2		// 2-radix
#define	R4	4		// 4-radix
//#define	PI	3.141592653589793238	

//-----------------------------------------------------------------------------
//---------------------------- GPU Kernels ------------------------------------
//-----------------------------------------------------------------------------
__global__ void FFT_Iter_R2(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	long long j = bx * blockDim.x + tx;	//Thread Id
	long long idxS = j;
	float v_r[R2]; float v_i[R2];		//2 inputs of butterfly
	//angle come from W = e^(-2*PI/(R2*Ns) * m)
	float angle = -2*PI*(j%Ns) / (Ns*R2);	//Ns=1 --> angle=0 , Ns=2 --> angle=0 & -2PI/4 , and so on...
	//Reading From Memory and multiplying in W
	float v_r_temp[R2]; float v_i_temp[R2];
	for (int r=0; r<R2; r++){
		v_r_temp[r] = x_r_d[idxS+r*N/R2];
		v_i_temp[r] = x_i_d[idxS+r*N/R2];		
		v_r[r] = (v_r_temp[r]*cos(r*angle) - v_i_temp[r]*sin(r*angle));	// (v_r + i v_i)*(cos + i sin) = (v_r*cos - v_i*sin) + i (v_r*sin + v_i*cos)
		v_i[r] = (v_r_temp[r]*sin(r*angle) + v_i_temp[r]*cos(r*angle));			
	}
	//Butterfly (R = 2)
	float v0_r = v_r[0]; float v0_i = v_i[0];
	v_r[0] = v0_r + v_r[1]; v_i[0] = v0_i + v_i[1];
	v_r[1] = v0_r - v_r[1]; v_i[1] = v0_i - v_i[1];
	//expand
	int idxD = (j/Ns)*Ns*R2 + (j%Ns);
	//Write in X_d
	for (int r=0; r<R2 ; r++){
		X_r_d[idxD + r*Ns] = v_r[r];
		X_i_d[idxD + r*Ns] = v_i[r];
	}
}

__global__ void FFT_Iter_R4(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	long long j = bx * blockDim.x + tx;	//Thread Id
	long long idxS = j;
	float v_r[R4]; float v_i[R4];		//2 inputs of butterfly
	//angle come from W = e^(-2*PI/(R4*Ns) * m)
	float angle = -2*PI*(j%Ns) / (Ns*R4);	//Ns=1 --> angle=0 , Ns=2 --> angle=0 & -2PI/4 , and so on...
	//Reading From Memory and multiplying in W
	float v_r_temp[R4]; float v_i_temp[R4];
	for (int r=0; r<R4; r++){
		v_r_temp[r] = x_r_d[idxS+r*N/R4];
		v_i_temp[r] = x_i_d[idxS+r*N/R4];		
		v_r[r] = (v_r_temp[r]*cos(r*angle) - v_i_temp[r]*sin(r*angle));	// (v_r + i v_i)*(cos + i sin) = (v_r*cos - v_i*sin) + i (v_r*sin + v_i*cos)
		v_i[r] = (v_r_temp[r]*sin(r*angle) + v_i_temp[r]*cos(r*angle));			
	}
	//Butterfly (R = 4)
	float v0_r = v_r[0]; float v0_i = v_i[0];
	float v1_r = v_r[1]; float v1_i = v_i[1];
	float v2_r = v_r[2]; float v2_i = v_i[2];
	float v3_r = v_r[3]; 
	v_r[0] = v0_r + v1_r + v2_r + v_r[3]; 	v_i[0] = v0_i + v1_i + v2_i + v_i[3];
	v_r[1] = v0_r + v1_i - v2_r - v_i[3];	v_i[1] = v0_i - v1_r - v2_i + v_r[3];
	v_r[2] = v0_r - v1_r + v2_r - v_r[3];	v_i[2] = v0_i - v1_i + v2_i - v_i[3];
	v_r[3] = v0_r - v1_i - v2_r + v_i[3];	v_i[3] = v0_i + v1_r - v2_i - v3_r;	
	//expand
	int idxD = (j/Ns)*Ns*R4 + (j%Ns);
	//Write in X_d
	for (int r=0; r<R4 ; r++){
		X_r_d[idxD + r*Ns] = v_r[r];
		X_i_d[idxD + r*Ns] = v_i[r];
	}
}

__global__ void Copy_X_to_x (float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d){	
	long long t_Id = bx * blockDim.x + tx;	//Thread Id	
	x_r_d[t_Id] = X_r_d[t_Id];
	x_i_d[t_Id] = X_i_d[t_Id];	
}

/* __global__ void FftIteration_eff(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	long long j = bx * blockDim.x + tx;	//Thread Id
	long long idxS = j;

	float v_r[R2]; float v_i[R2];		//2 inputs of butterfly
	//angle come from W = e^(-2*PI/(R2*Ns) * m)
	float angle = -2*PI*(j%Ns) / (Ns*R2);	//Ns=1 --> angle=0 , Ns=2 --> angle=0 & -2PI/4 , and so on...
	
	//Reading From Memory and multiplying in W
	float v_r_temp[R2]; float v_i_temp[R2];
	for (int r=0; r<R2; r++){
		v_r_temp[r] = x_r_d[idxS+r*N/R2];
		v_i_temp[r] = x_i_d[idxS+r*N/R2];
		
		v_r[r] = (v_r_temp[r]*cos(r*angle) - v_i_temp[r]*sin(r*angle));	// (v_r + i v_i)*(cos + i sin) = (v_r*cos - v_i*sin) + i (v_r*sin + v_i*cos)
		v_i[r] = (v_r_temp[r]*sin(r*angle) + v_i_temp[r]*cos(r*angle));	
		
	}
	
	//Butterfly (must changes for R2 that isnt equal 2)
	float v0_r = v_r[0]; float v0_i = v_i[0];
	v_r[0] = v0_r + v_r[1]; v_i[0] = v0_i + v_i[1];
	v_r[1] = v0_r - v_r[1]; v_i[1] = v0_i - v_i[1];
	
	//expand
	//int idxD = (j/Ns)*Ns*R2 + (j%Ns);
	
	//Write in X_d
	//for (int r=0; r<R2 ; r++){
	//	X_r_d[idxD + r*Ns] = v_r[r];
	//	X_i_d[idxD + r*Ns] = v_i[r];
	//}
	
	//--===Shared Memory for writing===--
	int idxD = (tx/Ns)^R2 + (tx%Ns);
	//Exchange
	
	//========
	idxD = bx* blockDim.x*R2 + tx;
	for (int r=0; r<R2; r++){
		X_r_d[idxD + r*blockDim.x] = v_r[r];
		X_i_d[idxD + r*blockDim.x] = v_i[r];
	}
} */
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//---------------------------- GPU Simple  ------------------------------------
//-----------------------------------------------------------------------------
void gpuKernel_simple(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{	
	if ( M<11 ){	
		for (int Ns=1; Ns<N ; Ns*=R2){							
			FFT_Iter_R2 <<< 1, N/R2 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads
			Copy_X_to_x <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
	}
	else {
		for (int Ns=1; Ns<N ; Ns*=R2){							
			FFT_Iter_R2 <<< N/(1024*R2), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads		//if .1 ms tasir dare
			Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
	}	
	
}




//-----------------------------------------------------------------------------
//---------------------------- GPU Efficient  ------------------------------------
//-----------------------------------------------------------------------------
void gpuKernel_efficient(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{	
	int Ns;
	if (M>24) {
		//========= M=25===========
		printf("\nM=25\n");
		// 1st Iteration
		Ns = 1;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 2nd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 3rd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 4th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 5th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 6th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 7th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 8th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 9th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 10th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 11th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 12th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 13th Iteration (2-radix)
		Ns = N/2;
		FFT_Iter_R2 <<< N/(1024*R2), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		
	}
	else if (M>23){
		//======= M=24 =======
		printf("\nM=24\n");
		// 1st Iteration
		Ns = 1;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 2nd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 3rd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 4th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 5th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 6th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 7th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 8th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 9th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 10th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 11th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 12th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// Copy x to X
		Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(X_r_d, X_i_d, x_r_d, x_i_d);
	}
	else if (M>22){
		//======= M=23 =======
		printf("\nM=23\n");
		// 1st Iteration
		Ns = 1;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 2nd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 3rd Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 4th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 5th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 6th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 7th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 8th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 9th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 10th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// 11th Iteration
		Ns = Ns*R4;
		FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
		// 12th Iteration
		Ns = N/2;
		FFT_Iter_R2 <<< N/(1024*R2), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		// copy x to X
		Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(X_r_d, X_i_d, x_r_d, x_i_d);
	}
	
	//========== Other Ms ========================================================
	else if ( M<11 ){
		if (M%2){
			for (Ns=1; Ns<(N/2) ; Ns*=R4){							
				FFT_Iter_R4 <<< 1, N/R4 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads
				Copy_X_to_x <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
			Ns = N/2;
			FFT_Iter_R2 <<< 1, N/R2 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads
			Copy_X_to_x <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
		else{
			for (int Ns=1; Ns<N ; Ns*=R4){							
			FFT_Iter_R4 <<< 1, N/R4 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads
			Copy_X_to_x <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
		}		
	}
	else {
		if (M%2){
			for (Ns=1; Ns<(N/2) ; Ns*=R4){							
				FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
			Ns = N/2;
			FFT_Iter_R2 <<< N/(1024*R2), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);	//blockDim.x = 2^10 , gridDim.x = 2^(M-11) , N/2 Threads
			Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
		else {
			for (Ns=1; Ns<N ; Ns*=R4){							
				FFT_Iter_R4 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				Copy_X_to_x <<< (1<<M-10), (1<<10) >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
		}
	}
}
